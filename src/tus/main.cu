#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <iostream>

#include "physics.h"
#include "csv.h"
#include "helper.h"
#include "data_t.h"
#include "constant.h"
#include "basic_kernel.h"
#include <cassert>

// Comment out this line to enable debug mode
// #define NDEBUG

#define SIM_TIME 10
#define STEP_SIZE 1


int main(int argc, char *argv[])
{
    // leave this here just for simple reference 
    //CORE::VEL v{1.0f, 2.0f, 3.0f};
    //v *= 2.0f;
    //std::cout << v << std::endl;

    /* Get Dimension */
    /// TODO: Add more arguments for input and output
    /// Haiqi: I think it should be "main [num_body] [simulation_end_time] [num_iteration] or [step_size]". or we simply let step_size = 1
    if (argc != 3)
    {
        printf("Error: The number of arguments is %d, but not exactly 2\n", argc);
        return 0;
    }

    /* CSV files of initial conditions */
    unsigned nBody = atoi(argv[1]);
    std::string csv_path(argv[2]);
    // temporarily assign them to MARCO
    unsigned simulation_time = SIM_TIME;
    unsigned step_size = STEP_SIZE;
    
    /* CSV files of initial conditions */

    auto ic = CORE::parse_body_ic_from_csv(csv_path); 
    
    // TODO: get better debug message.
    assert(ic.size() == nBody);

    // random initializer just for now
    srand(time(NULL));
    size_t vector_size = sizeof(data_t_3d) * nBody;
    size_t data_size = sizeof(data_t) * nBody;

    /*
     *   host side memory allocation
     */
    data_t_3d *h_X, *h_A, *h_V, *h_output_X;
    data_t *h_M;
    host_malloc_helper((void **)&h_X, vector_size);
    host_malloc_helper((void **)&h_A, vector_size);
    host_malloc_helper((void **)&h_V, vector_size);
    host_malloc_helper((void **)&h_output_X, vector_size);
    host_malloc_helper((void **)&h_M, data_size);

    /*
     *   input randome initialize
     */
    
    parse_ic(h_V, h_X, ic);
    
    /*
     *   input randome initialize
     */
    random_initialize_mass(h_M, nBody, RANDOM_RANGE);

    /*
     *  mass 
     */
    data_t *d_M;
    gpuErrchk(hipMalloc((void **)&d_M, data_size));
    /*
     *   create double buffer on device side
     */
    data_t_3d **d_X, **d_A, **d_V;
    unsigned src_index = 0;
    unsigned dest_index = 1;
    d_X = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
    gpuErrchk(hipMalloc((void **)&d_X[src_index], vector_size));
    gpuErrchk(hipMalloc((void **)&d_X[dest_index], vector_size));

    d_A = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
    gpuErrchk(hipMalloc((void **)&d_A[src_index], vector_size));
    gpuErrchk(hipMalloc((void **)&d_A[dest_index], vector_size));

    d_V = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
    gpuErrchk(hipMalloc((void **)&d_V[src_index], vector_size));
    gpuErrchk(hipMalloc((void **)&d_V[dest_index], vector_size));

    /*
     *   create double buffer on device side
     */
    // hipMemcpy(d_A[0], h_A, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_X[src_index], h_X, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_V[src_index], h_V, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, data_size, hipMemcpyHostToDevice);

    unsigned nthreads = 256;
    unsigned nblocks = (nBody + nthreads - 1) / nthreads;

    // calculate the initialia acceleration
    calculate_acceleration<<<nblocks, nthreads>>>(nBody, d_X[src_index], d_M, d_A[src_index]);
	
    std::cout << "Start Computation\n";

    for (unsigned step = 0; step < simulation_time; step += step_size)
    {

        // There should be more than one ways to do synchronization. I temporarily randomly choosed one
        calculate_acceleration<<<nblocks, nthreads>>>(nBody, d_X[src_index], d_M,                                                          //input
                                                      d_A[dest_index]);                                                                    // output
        update_step<<<nblocks, nthreads>>>(nBody, (data_t)step_size, d_X[src_index], d_V[src_index], d_A[src_index], d_M, d_A[dest_index], //input
                                           d_X[dest_index], d_V[dest_index]);                                                              // output
	
	// we don't have to synchronize here but this gices a better visualization on how fast / slow the program is 	
	std::cout << "epoch " << step << std::endl;
	hipDeviceSynchronize();

        swap(src_index, dest_index);
    }
    hipDeviceSynchronize();
    std::cout << "Finished Compuation\n";
    // at end, the final data is actually at src_index because the last swap
    hipMemcpy(h_output_X, d_X[src_index], vector_size, hipMemcpyDeviceToHost);

    // Just for debug purpose on small inputs
    for (unsigned i = 0; i < nBody; i++)
    {
        //printf("object = %d, %f, %f, %f\n", i, h_output_X[i].x, h_output_X[i].y, h_output_X[i].z);
    }

    return 0;
}
