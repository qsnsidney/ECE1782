#include "hip/hip_runtime.h"
#include "nvda_improved_engine.cuh"
#include "core/timer.h"

#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <iostream>

#include "core/physics.hpp"
#include "core/serde.h"
#include "helper.cuh"
#include "data_t.cuh"
#include "nvda_improved_kernel.cuh"

namespace
{
    CORE::SYSTEM_STATE generate_system_state(const float4 *h_X, const data_t_3d *h_V, const size_t nbody)
    {
        CORE::SYSTEM_STATE system_state;
        system_state.reserve(nbody);
        for (size_t i_body = 0; i_body < nbody; i_body++)
        {
            CORE::POS pos_temp{h_X[i_body].x, h_X[i_body].y, h_X[i_body].z};
            CORE::VEL vel_temp{h_V[i_body].x, h_V[i_body].y, h_V[i_body].z};
            system_state.emplace_back(pos_temp, vel_temp, h_X[i_body].w);
        }
        return system_state;
    }
}

namespace TUS
{
    NVDA_IMPROVED_ENGINE::NVDA_IMPROVED_ENGINE(CORE::SYSTEM_STATE system_state_ic,
                                       CORE::DT dt,
                                       int block_size,
                                       std::optional<std::string> system_state_log_dir_opt) : ENGINE(std::move(system_state_ic), dt, std::move(system_state_log_dir_opt)),
                                                                                              block_size_(block_size)
    {
    }

    CORE::SYSTEM_STATE NVDA_IMPROVED_ENGINE::execute(int n_iter, CORE::TIMER &timer)
    {
        size_t nBody = system_state_snapshot().size();

        /* BIN file of initial conditions */
        const auto &ic = system_state_snapshot();

        // random initializer just for now
        size_t vector_size_3d = sizeof(data_t_3d) * nBody;
        size_t vector_size_4d = sizeof(float4) * nBody;
        size_t vector_size_4dx = sizeof(float4) * ((nBody + (block_size_ - 1))/block_size_) * block_size_;
        /*
     *   host side memory allocation
     */
        data_t_3d *h_V, *h_output_V;
        float4 *h_X, *h_A, *h_output_X;

        host_malloc_helper((void **)&h_V, vector_size_3d);
        host_malloc_helper((void **)&h_output_V, vector_size_3d);

        host_malloc_helper((void **)&h_X, vector_size_4dx);
        host_malloc_helper((void **)&h_A, vector_size_4d);
        host_malloc_helper((void **)&h_output_X, vector_size_4d);

        timer.elapsed_previous("allocated host side memory");
        /*
     *   input randome initialize
     */

        for(int i = 0; i < ((nBody + (block_size_ - 1))/block_size_) * block_size_; i++) {
            h_X[i] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        }

        parse_ic_f4(h_X, h_V, ic);
        timer.elapsed_previous("deserialize_body_state_vec_from_csv");

        /*
         * create double buffer on device side
         */
        float4 **d_X, **d_A;
        unsigned src_index = 0;
        unsigned dest_index = 1;
        d_X = (float4 **)malloc(2 * sizeof(float4 *));
        gpuErrchk(hipMalloc((void **)&d_X[src_index], vector_size_4dx));
        gpuErrchk(hipMalloc((void **)&d_X[dest_index], vector_size_4dx));

        d_A = (float4 **)malloc(2 * sizeof(float4 *));
        gpuErrchk(hipMalloc((void **)&d_A[src_index], vector_size_4d));
        gpuErrchk(hipMalloc((void **)&d_A[dest_index], vector_size_4d));

        data_t_3d **d_V;
        d_V = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
        gpuErrchk(hipMalloc((void **)&d_V[src_index], vector_size_3d));
        gpuErrchk(hipMalloc((void **)&d_V[dest_index], vector_size_3d));

        data_t_3d *d_V_half;
        gpuErrchk(hipMalloc((void **)&d_V_half, vector_size_3d));

        timer.elapsed_previous("allocated device memory");
        /*
         *   create double buffer on device side
         */
        // hipMemcpy(d_A[0], h_A, vector_size, hipMemcpyHostToDevice);
        hipMemcpy(d_X[src_index], h_X, vector_size_4dx, hipMemcpyHostToDevice);
        hipMemcpy(d_V[src_index], h_V, vector_size_3d, hipMemcpyHostToDevice);
        timer.elapsed_previous("copied input data from host to device");

        // nthread is assigned to either 32 by default or set to a custom power of 2 by user
        std::cout << "Set thread_per_block to " << block_size_ << std::endl;
        unsigned nblocks = (nBody + block_size_ - 1) / block_size_;

        // calculate the initialia acceleration
        calculate_forces_improved<<<nblocks, block_size_, block_size_ * sizeof(float4)>>>(nBody, d_X[src_index], d_A[src_index], block_size_);
        timer.elapsed_previous("Calculated initial acceleration");

        {
            CORE::TIMER core_timer("all_iters");
            for (int i_iter = 0; i_iter < n_iter; i_iter++)
            {
                update_step_pos_f4<<<nblocks, block_size_>>>(nBody, (data_t)dt(), d_X[src_index], d_V[src_index], d_A[src_index], //input
                                                            d_X[dest_index], d_V_half);                                               // output

                hipDeviceSynchronize();

                calculate_forces_improved<<<nblocks, block_size_, block_size_ * sizeof(float4)>>>(nBody, d_X[dest_index], //input
                                                                   d_A[dest_index], block_size_);            // output

                hipDeviceSynchronize();

                update_step_vel_f4<<<nblocks, block_size_>>>(nBody, (data_t)dt(), d_A[dest_index], d_V_half, //input
                                                            d_V[dest_index]);                                    // output
                hipDeviceSynchronize();

                timer.elapsed_previous(std::string("iter") + std::to_string(i_iter), CORE::TIMER::TRIGGER_LEVEL::INFO);

                if (is_system_state_logging_enabled())
                {
                    hipMemcpy(h_output_X, d_X[dest_index], vector_size_4d, hipMemcpyDeviceToHost);
                    hipMemcpy(h_output_V, d_V[dest_index], vector_size_4d, hipMemcpyDeviceToHost);

                    if (i_iter == 0)
                    {
                        push_system_state_to_log(generate_system_state(h_X, h_V, nBody));
                    }
                    push_system_state_to_log(generate_system_state(h_output_X, h_output_V, nBody));

                    if (i_iter % 10 == 0)
                    {
                        serialize_system_state_log();
                    }

                    timer.elapsed_previous(std::string("Transfer to CPU"), CORE::TIMER::TRIGGER_LEVEL::INFO);
                }

                swap(src_index, dest_index);
            }
            hipDeviceSynchronize();
        }

        // at end, the final data is actually at src_index because the last swap
        hipMemcpy(h_output_X, d_X[src_index], vector_size_4d, hipMemcpyDeviceToHost);
        hipMemcpy(h_output_V, d_V[src_index], vector_size_3d, hipMemcpyDeviceToHost);

        // Hack Hack Hack. dump out the data
        hipMemcpy(h_A, d_A[src_index], vector_size_4d, hipMemcpyDeviceToHost);

        write_debug_output(name(), h_output_X, h_output_V, h_A, nBody);

        timer.elapsed_previous("copied output back to host");

        //Just for debug purpose on small inputs
        // for (unsigned i = 0; i < nBody; i++)
        // {
        //    printf("object = %d, %f, %f, %f\n", i, h_output_X[i].x, h_output_X[i].y, h_output_X[i].z);
        // }

        auto system_state_result = generate_system_state(h_output_X, h_output_V, nBody);

        hipHostFree(h_X);
        hipHostFree(h_A);
        hipHostFree(h_V);
        hipHostFree(h_output_X);
        hipHostFree(h_output_V);

        for(const auto i : {src_index, dest_index}){
            hipFree(d_X[i]);
            hipFree(d_V[i]);
            hipFree(d_A[i]);
        }
        hipFree(d_V_half);
        hipDeviceReset();

        return system_state_result;
    }
}