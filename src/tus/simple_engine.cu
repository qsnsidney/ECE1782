#include "simple_engine.cuh"
#include "core/timer.h"

#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <iostream>

#include "core/physics.hpp"
#include "core/serde.h"
#include "helper.cuh"
#include "data_t.cuh"
#include "constant.h"
#include "basic_kernel.cuh"

namespace {
    CORE::BODY_STATE_VEC generate_body_state_vec(const data_t_3d *h_X, const data_t_3d *h_V, const data_t *mass, const size_t nbody)
    {
        CORE::BODY_STATE_VEC body_states;
        body_states.reserve(nbody);
        for (size_t i_body = 0; i_body < nbody; i_body++)
        {
            CORE::POS pos_temp{h_X[i_body].x, h_X[i_body].y, h_X[i_body].z};
            CORE::VEL vel_temp{h_V[i_body].x, h_V[i_body].y, h_V[i_body].z};
            body_states.emplace_back(pos_temp, vel_temp, mass[i_body]);
        }
        return body_states;
    }
}

namespace TUS
{
    SIMPLE_ENGINE::SIMPLE_ENGINE(CORE::BODY_STATE_VEC body_states_ic,
                                 CORE::DT dt,
                                 int block_size,
                                 std::optional<std::string> body_states_log_dir_opt) : ENGINE(std::move(body_states_ic), dt, std::move(body_states_log_dir_opt)),
                                                                                       block_size_(block_size)
    {
    }

    CORE::BODY_STATE_VEC SIMPLE_ENGINE::execute(int n_iter)
    {
        size_t nBody = body_states_ic().size();

        CORE::TIMER timer(std::string("SIMPLE_ENGINE(") + std::to_string(nBody) + "," + std::to_string(dt()) + "*" + std::to_string(n_iter) + ")");

        /* BIN file of initial conditions */
        const auto &ic = body_states_ic();

        // random initializer just for now
        srand(time(NULL));
        size_t vector_size = sizeof(data_t_3d) * nBody;
        size_t data_size = sizeof(data_t) * nBody;

        /*
     *   host side memory allocation
     */
        data_t_3d *h_X, *h_A, *h_V, *h_output_X, *h_output_V;
        data_t *h_M;
        host_malloc_helper((void **)&h_X, vector_size);
        host_malloc_helper((void **)&h_A, vector_size);
        host_malloc_helper((void **)&h_V, vector_size);
        host_malloc_helper((void **)&h_output_X, vector_size);
        host_malloc_helper((void **)&h_output_V, vector_size);
        host_malloc_helper((void **)&h_M, data_size);
        timer.elapsed_previous("allocated host side memory");
        /*
     *   input randome initialize
     */

        parse_ic(h_X, h_V, h_M, ic);
        timer.elapsed_previous("deserialize_body_state_vec_from_csv");

        /*
     *  mass 
     */
        data_t *d_M;
        gpuErrchk(hipMalloc((void **)&d_M, data_size));
        /*
     *   create double buffer on device side
     */
        data_t_3d **d_X, **d_A, **d_V;
        unsigned src_index = 0;
        unsigned dest_index = 1;
        d_X = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
        gpuErrchk(hipMalloc((void **)&d_X[src_index], vector_size));
        gpuErrchk(hipMalloc((void **)&d_X[dest_index], vector_size));

        d_A = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
        gpuErrchk(hipMalloc((void **)&d_A[src_index], vector_size));
        gpuErrchk(hipMalloc((void **)&d_A[dest_index], vector_size));

        d_V = (data_t_3d **)malloc(2 * sizeof(data_t_3d *));
        gpuErrchk(hipMalloc((void **)&d_V[src_index], vector_size));
        gpuErrchk(hipMalloc((void **)&d_V[dest_index], vector_size));

        data_t_3d *d_V_half;
        gpuErrchk(hipMalloc((void **)&d_V_half, vector_size));

        timer.elapsed_previous("allocated device memory");
        /*
     *   create double buffer on device side
     */
        // hipMemcpy(d_A[0], h_A, vector_size, hipMemcpyHostToDevice);
        hipMemcpy(d_X[src_index], h_X, vector_size, hipMemcpyHostToDevice);
        hipMemcpy(d_V[src_index], h_V, vector_size, hipMemcpyHostToDevice);
        hipMemcpy(d_M, h_M, data_size, hipMemcpyHostToDevice);
        timer.elapsed_previous("copied input data from host to device");

        // nthread is assigned to either 32 by default or set to a custom power of 2 by user
        std::cout << "Set thread_per_block to " << block_size_ << std::endl;
        unsigned nblocks = (nBody + block_size_ - 1) / block_size_;

        // calculate the initialia acceleration
        calculate_acceleration<<<nblocks, block_size_>>>(nBody, d_X[src_index], d_M, d_A[src_index]);
        timer.elapsed_previous("Calculated initial acceleration");

        {
            CORE::TIMER core_timer("computation_core");
            for (int i_iter = 0; i_iter < n_iter; i_iter++)
            {
                update_step_pos<<<nblocks, block_size_>>>(nBody, (data_t)dt(), d_X[src_index], d_V[src_index], d_A[src_index], d_M, //input
                                                            d_X[dest_index], d_V_half); // output

                hipDeviceSynchronize();

                calculate_acceleration<<<nblocks, block_size_>>>(nBody, d_X[dest_index], d_M, //input
                                                                   d_A[dest_index]); // output

                hipDeviceSynchronize();

                update_step_vel<<<nblocks, block_size_>>>(nBody, (data_t)dt(), d_M, d_A[dest_index], d_V_half, //input
                                                            d_V[dest_index]); // output
                hipDeviceSynchronize();

                timer.elapsed_previous(std::string("iter") + std::to_string(i_iter));

                if (is_body_states_logging_enabled())
                {
                    hipMemcpy(h_output_X, d_X[dest_index], vector_size, hipMemcpyDeviceToHost);
                    hipMemcpy(h_output_V, d_V[dest_index], vector_size, hipMemcpyDeviceToHost);

                    if (i_iter == 0)
                    {
                        push_body_states_to_log(generate_body_state_vec(h_X, h_V, h_M, nBody));
                    }
                    push_body_states_to_log(generate_body_state_vec(h_output_X, h_output_V, h_M, nBody));

                    if (i_iter % 10 == 0)
                    {
                        serialize_body_states_log();
                    }

                    timer.elapsed_previous(std::string("Transfer to CPU"));
                }

                swap(src_index, dest_index);
            }
            hipDeviceSynchronize();
        }

        // at end, the final data is actually at src_index because the last swap
        hipMemcpy(h_output_X, d_X[src_index], vector_size, hipMemcpyDeviceToHost);
        hipMemcpy(h_output_V, d_V[src_index], vector_size, hipMemcpyDeviceToHost);
        timer.elapsed_previous("copied output back to host");
        
        // Just for debug purpose on small inputs
        // for (unsigned i = 0; i < nBody; i++)
        // {
        //    //printf("object = %d, %f, %f, %f\n", i, h_output_X[i].x, h_output_X[i].y, h_output_X[i].z);
        // }

        return generate_body_state_vec(h_output_X, h_output_V, h_M, nBody);
    }
}